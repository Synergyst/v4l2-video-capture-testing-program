#include "hip/hip_runtime.h"
// vignette_cuda.cu
#include "vignette_cuda.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

// Simple CUDA error helper
static inline bool check_cuda(const char* msg, hipError_t err) {
  if (err != hipSuccess) {
    fprintf(stderr, "[cuda][vignette] %s: %s\n", msg, hipGetErrorString(err));
    return false;
  }
  return true;
}

// Reused device buffer to avoid reallocations each frame
static uint8_t* g_d_rgb = nullptr;
static size_t   g_d_capacity = 0; // bytes
static int      g_last_w = 0;
static int      g_last_h = 0;

__global__ void vignette_kernel(uint8_t* d_rgb, int width, int height, VignetteParamsCUDA p) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int total = width * height;
  if (idx >= total) return;

  int x = idx % width;
  int y = idx / width;

  // Normalize coords to [-0.5..0.5]-ish range using image dims
  float cx = p.center_x * (float)width;
  float cy = p.center_y * (float)height;
  float nx = (x - cx) / (float)width;
  float ny = (y - cy) / (float)height;

  // Rotate by angle (compute sin/cos once per thread; cheap vs memory traffic)
  float s = sinf(p.angle_rad);
  float c = cosf(p.angle_rad);
  float rx = c * nx - s * ny;
  float ry = s * nx + c * ny;

  // Elliptical scaling
  rx *= p.axis_scale_x;
  ry *= p.axis_scale_y;

  // Radial distance
  float r = sqrtf(rx * rx + ry * ry);

  // Map radius to [0..1] falloff coordinate t
  float inner = p.inner_radius;
  float outer = p.outer_radius;
  float denom = fmaxf(outer - inner, 1e-6f);
  float t = (r - inner) / denom;
  t = fminf(fmaxf(t, 0.0f), 1.0f);

  // Smoothstep falloff and apply strength
  float w = t * t * (3.f - 2.f * t);
  float gain = 1.f - p.strength * w;
  if (gain < 0.f) gain = 0.f;

  int base = idx * 3;
  float r0 = d_rgb[base + 0] * gain;
  float g0 = d_rgb[base + 1] * gain;
  float b0 = d_rgb[base + 2] * gain;

  // Clamp and write back
  d_rgb[base + 0] = (uint8_t)fminf(fmaxf(r0, 0.f), 255.f);
  d_rgb[base + 1] = (uint8_t)fminf(fmaxf(g0, 0.f), 255.f);
  d_rgb[base + 2] = (uint8_t)fminf(fmaxf(b0, 0.f), 255.f);
}

extern "C" bool apply_vignette_cuda(uint8_t* host_rgb, int width, int height, const VignetteParamsCUDA* params) {
  if (!host_rgb || !params || width <= 0 || height <= 0) return false;

  // Ensure CUDA context
  if (!check_cuda("hipFree(0) init", hipFree(0))) return false;

  size_t bytes = (size_t)width * (size_t)height * 3ull;
  if (g_d_capacity < bytes || width != g_last_w || height != g_last_h) {
    if (g_d_rgb) hipFree(g_d_rgb);
    g_d_rgb = nullptr;
    g_d_capacity = 0;
    if (!check_cuda("hipMalloc d_rgb", hipMalloc((void**)&g_d_rgb, bytes))) return false;
    g_d_capacity = bytes;
    g_last_w = width;
    g_last_h = height;
  }

  // H2D
  if (!check_cuda("hipMemcpy H2D", hipMemcpy(g_d_rgb, host_rgb, bytes, hipMemcpyHostToDevice))) return false;

  // Launch kernel
  int threads = 256;
  int pixels  = width * height;
  int blocks  = (pixels + threads - 1) / threads;
  vignette_kernel<<<blocks, threads>>>(g_d_rgb, width, height, *params);
  if (!check_cuda("kernel launch", hipGetLastError())) return false;
  if (!check_cuda("hipDeviceSynchronize", hipDeviceSynchronize())) return false;

  // D2H
  if (!check_cuda("hipMemcpy D2H", hipMemcpy(host_rgb, g_d_rgb, bytes, hipMemcpyDeviceToHost))) return false;

  return true;
}
