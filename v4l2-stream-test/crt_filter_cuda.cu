#include "hip/hip_runtime.h"
// crt_filter.cu
#include "crt_filter_cuda.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <algorithm>
#include <random>

// ---------------- CUDA helpers ----------------
static inline bool check_cuda(const char* msg, hipError_t err) {
  if (err != hipSuccess) {
    fprintf(stderr, "[cuda][CRT] %s: %s\n", msg, hipGetErrorString(err));
    return false;
  }
  return true;
}

// Persistent device buffers reused across frames.
// Note: this is module-scoped (shared if you ever make multiple CRTFilter instances).
static uint8_t* g_d_src = nullptr;
static uint8_t* g_d_dst = nullptr;
static size_t   g_d_capacity = 0; // bytes
static int      g_last_w = 0;
static int      g_last_h = 0;

// ---------------- Per-frame constants passed to kernel ----------------
struct CRTKernelParams {
  // image info
  int   width;
  int   height;
  // per-frame constants
  float t;           // seconds
  float flicker;     // overall brightness modulator
  int   v_shift_pix; // vertical shift in lines (rounded)
  int   vphase;      // 0/1 (scanline alternation phase)
  int   frame_idx;   // frame counter (for hashing)
  // fixed params from CRTParams
  float scanline_strength;  // 0..1
  float mask_strength;      // 0..1 (already used to build tri gains)
  float grain_strength;     // amplitude multiplier
  float h_warp_amp;         // pixels
  float h_warp_freq_y;      // per-line
  float h_warp_freq_t;      // per-second
  float wobble_line_noise;  // pixels
  float phaseH;             // random phase for horizontal wobble
  // triad shadow-mask multipliers (only the mask part; base_gain is per-line)
  float triR[3];
  float triG[3];
  float triB[3];
};

// Device-side helpers
__device__ inline uint8_t clamp_u8_dev(int v) {
  return (uint8_t)(v < 0 ? 0 : (v > 255 ? 255 : v));
}
// hash like your host-side version: [-1,1]
__device__ inline float hash31_dev(int x, int y, int t) {
  // uint32_t h = (uint32_t)(x * 374761393u + y * 668265263u) ^ (uint32_t)(t * 362437u);
  unsigned int h = (unsigned int)(x * 374761393u + y * 668265263u) ^ (unsigned int)(t * 362437u);
  h = (h ^ (h >> 13)) * 1274126177u;
  h ^= (h >> 16);
  // map to [-1, 1]
  float f = (float)(h & 0x7FFFFFu) / (float)0x3FFFFFu; // ~[0,2)
  return f * 2.0f - 1.0f;
}

__global__ void crt_kernel(const uint8_t* __restrict__ src,
                           uint8_t* __restrict__ dst,
                           CRTKernelParams kp)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int total = kp.width * kp.height;
  if (idx >= total) return;

  const int x = idx % kp.width;
  const int y = idx / kp.width;

  // Vertical source line with jitter/bounce (already rounded on host)
  int y_src = y - kp.v_shift_pix;
  if (y_src < 0)           y_src = 0;
  if (y_src >= kp.height)  y_src = kp.height - 1;

  // Horizontal wobble (constant across line) + per-line noise
  // wobble_base = h_warp_amp * sin(2*pi*(h_warp_freq_y*y + h_warp_freq_t*t) + phaseH)
  const float TWO_PI = 6.2831853071795864769f;
  float wobble_base = kp.h_warp_amp *
                      __sinf(TWO_PI * (kp.h_warp_freq_y * (float)y + kp.h_warp_freq_t * kp.t) + kp.phaseH);
  float wobble_noise = kp.wobble_line_noise * hash31_dev(0, y, kp.frame_idx);
  int wob_i = (int)nearbyintf(wobble_base + wobble_noise);

  // Scanline darkening (alternating lines)
  float scan_gain = (((y + kp.vphase) & 1) ? (1.0f - kp.scanline_strength) : 1.0f);
  float base_gain = kp.flicker * scan_gain;

  // Shadow-mask triad selection
  int tri = x % 3;
  float mr = base_gain * kp.triR[tri];
  float mg = base_gain * kp.triG[tri];
  float mb = base_gain * kp.triB[tri];

  // Source pixel with horizontal wobble, clamped
  int xs = x + wob_i;
  if (xs < 0)         xs = 0;
  if (xs >= kp.width) xs = kp.width - 1;

  int src_ofs = (y_src * kp.width + xs) * 3;
  int dst_ofs = idx * 3;

  // Grain per pixel
  float grain = kp.grain_strength * hash31_dev(x, y, kp.frame_idx);

  int r = (int)lrintf(src[src_ofs + 0] * mr + 127.5f * grain);
  int g = (int)lrintf(src[src_ofs + 1] * mg + 127.5f * grain);
  int b = (int)lrintf(src[src_ofs + 2] * mb + 127.5f * grain);

  dst[dst_ofs + 0] = clamp_u8_dev(r);
  dst[dst_ofs + 1] = clamp_u8_dev(g);
  dst[dst_ofs + 2] = clamp_u8_dev(b);
}

// ---------------- CRTFilter (host) ----------------
CRTFilter::CRTFilter(int width, int height, const CRTParams& p, size_t threads, int fps)
  : w_(width), h_(height), p_(p), pool_(threads), fps_((fps > 0) ? fps : 30) {
  std::random_device rd;
  rng_.seed(rd());
  const float TWO_PI = 6.2831853071795864769f;
  phase60_ = frand_(rng_, 0.0f, TWO_PI);
  phaseH_  = frand_(rng_, 0.0f, TWO_PI);
  phaseV_  = frand_(rng_, 0.0f, TWO_PI);
  // Create CUDA context in this thread
  (void)check_cuda("hipFree(0) init", hipFree(0));
}

CRTFilter::FrameConsts CRTFilter::prepare_frame_(int frame_idx, int fps) {
  const float TWO_PI = 6.2831853071795864769f;
  const float t = (fps > 0) ? (float)frame_idx / (float)fps : (float)frame_idx * (1.0f / 30.0f);

  // Per-frame flicker
  float flicker = 1.0f
    + p_.flicker_60hz * std::sin(TWO_PI * 60.0f * t + phase60_)
    + frand_(rng_, -p_.flicker_noise, p_.flicker_noise);

  // Occasional vertical sync "bounce"
  if (v_bounce_frames_ == 0 && frand_(rng_, 0.0f, 1.0f) < 0.015f) {
    v_bounce_frames_ = 50 + (int)frand_(rng_, 0.0f, 40.0f);
    v_bounce_amp_    = frand_(rng_, 1.0f, 3.0f);
  }

  float v_bounce = 0.0f;
  if (v_bounce_frames_ > 0) {
    float decay = (float)v_bounce_frames_ / 90.0f;
    v_bounce = v_bounce_amp_ * std::sin(TWO_PI * 2.0f * t + phaseV_) * decay;
    v_bounce_frames_--;
  }

  // Base vertical jitter
  float v_jitter = p_.v_shake_amp * std::sin(TWO_PI * 0.9f * t + phaseV_);
  float v_shift = v_bounce + v_jitter; // in lines

  FrameConsts fc;
  fc.t = t;
  fc.flicker = flicker;
  fc.v_shift = v_shift;
  fc.vphase = frame_idx & 1;
  return fc;
}

// Auto-advancing apply
void CRTFilter::apply(const uint8_t* src, std::vector<uint8_t>& dst) {
  this->apply(src, static_cast<int>(frame_idx_), fps_, dst);
  ++frame_idx_;
}

void CRTFilter::apply(const uint8_t* src, int frame_idx, int fps, std::vector<uint8_t>& dst) {
  if (!src || w_ <= 0 || h_ <= 0) {
    dst.clear();
    return;
  }
  dst.resize((size_t)w_ * (size_t)h_ * 3u);

  // Per-frame constants (host)
  const auto fc = prepare_frame_(frame_idx, fps);

  // Prepare kernel params
  CRTKernelParams kp{};
  kp.width  = w_;
  kp.height = h_;
  kp.t = fc.t;
  kp.flicker = fc.flicker;
  kp.v_shift_pix = (int)lrintf(fc.v_shift);
  kp.vphase = fc.vphase;
  kp.frame_idx = frame_idx;

  kp.scanline_strength = std::clamp(p_.scanline_strength, 0.0f, 1.0f);
  kp.mask_strength     = std::clamp(p_.mask_strength, 0.0f, 1.0f);
  kp.grain_strength    = std::max(0.0f, p_.grain_strength);
  kp.h_warp_amp        = p_.h_warp_amp;
  kp.h_warp_freq_y     = p_.h_warp_freq_y;
  kp.h_warp_freq_t     = p_.h_warp_freq_t;
  kp.wobble_line_noise = p_.wobble_line_noise;
  kp.phaseH            = phaseH_;

  // Triad mask gains (mask only; base_gain multiplies in kernel)
  const float ms = kp.mask_strength;
  const float triad[3][3] = {
      {1.00f, 0.80f, 0.75f}, // x%3 == 0
      {0.75f, 1.00f, 0.80f}, // x%3 == 1
      {0.80f, 0.75f, 1.00f}, // x%3 == 2
  };
  for (int i = 0; i < 3; ++i) {
    kp.triR[i] = (1.0f - ms) + ms * triad[i][0];
    kp.triG[i] = (1.0f - ms) + ms * triad[i][1];
    kp.triB[i] = (1.0f - ms) + ms * triad[i][2];
  }

  // Ensure device buffers are allocated
  const size_t bytes = (size_t)w_ * (size_t)h_ * 3ull;
  if (g_d_capacity < bytes || w_ != g_last_w || h_ != g_last_h) {
    if (g_d_src) hipFree(g_d_src);
    if (g_d_dst) hipFree(g_d_dst);
    g_d_src = g_d_dst = nullptr;
    g_d_capacity = 0;
    if (!check_cuda("hipMalloc d_src", hipMalloc((void**)&g_d_src, bytes))) return;
    if (!check_cuda("hipMalloc d_dst", hipMalloc((void**)&g_d_dst, bytes))) { hipFree(g_d_src); g_d_src = nullptr; return; }
    g_d_capacity = bytes;
    g_last_w = w_;
    g_last_h = h_;
  }

  // H2D
  if (!check_cuda("hipMemcpy H2D(src)", hipMemcpy(g_d_src, src, bytes, hipMemcpyHostToDevice))) return;

  // Launch kernel
  int threads = 256;
  int blocks = (int)((bytes / 3 + threads - 1) / threads); // one thread per pixel
  crt_kernel<<<blocks, threads>>>(g_d_src, g_d_dst, kp);
  if (!check_cuda("kernel launch", hipGetLastError())) return;
  if (!check_cuda("hipDeviceSynchronize", hipDeviceSynchronize())) return;

  // D2H
  if (!check_cuda("hipMemcpy D2H(dst)", hipMemcpy(dst.data(), g_d_dst, bytes, hipMemcpyDeviceToHost))) return;
}
